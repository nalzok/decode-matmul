#include "hip/hip_runtime.h"
#include <iostream>
#include <hip/hip_runtime.h>
#include <mma.h>

#include <ATen/ATen.h>
#include <ATen/Context.h>
#include <ATen/cuda/HIPContext.h>
#include <ATen/Dispatch.h>

#include <torch/types.h>
#include <torch/extension.h>

using namespace nvcuda;

#define CHECK_CUDA(x)           TORCH_CHECK(x.is_cuda(), #x " must be a CUDA tensor")
#define CHECK_CONTIGUOUS(x)     TORCH_CHECK(x.is_contiguous(), #x " must be contiguous")
#define CHECK_INPUT(x) 	        do { CHECK_CUDA(x); CHECK_CONTIGUOUS(x); } while(false)
#define gpuErrchk(ans)          do { gpuAssert((ans), __FILE__, __LINE__); } while (false)

#define WMMA_M                  8
#define WMMA_N                  32
#define WMMA_K                  16

#define WARP_SIZE               32

#define SMEM_SIZE_MAX           (48 * 1024)


__host__ static inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
    if (code != hipSuccess)
    {
        fprintf(stderr, "GPUassert[%s:%d]: %s\n", file, line, hipGetErrorString(code));
        if (abort) exit(code);
    }
}


__global__ static void decode_matmul_kernel(
        int32_t *__restrict__ output,
        const int8_t *__restrict__ x,
        const int16_t *__restrict__ weights_compressed,
        const int64_t *__restrict__ codebook_abs,
        const int64_t *__restrict__ codebook_sign,
        int64_t GLOBAL_M,
        int64_t GLOBAL_N,
        int64_t GLOBAL_K) {

    int64_t warpId = threadIdx.x / WARP_SIZE;
    int64_t laneId = threadIdx.x % WARP_SIZE;

    extern __shared__ __align__(sizeof(int64_t)) char raw[];
    int64_t *decoded_block = reinterpret_cast<int64_t *>(raw);
    int64_t *decoded = decoded_block + warpId * WMMA_N * WMMA_K/8;

    int64_t offset = blockDim.x/WARP_SIZE * WMMA_N * WMMA_K/8 * sizeof(int64_t);
    int32_t *result_block = reinterpret_cast<int32_t *>(raw + offset);
    int32_t *result = result_block + warpId * WMMA_M * WMMA_N;
    for (int64_t i = laneId; i < WMMA_M * WMMA_N; i += WARP_SIZE) {
        result[i] = 0;
    }

    int64_t TILES_M = GLOBAL_M / WMMA_M;
    int64_t TILES_N = GLOBAL_N / WMMA_N;
    int64_t TILES_K = GLOBAL_K / WMMA_K;

    wmma::fragment<wmma::matrix_a, WMMA_M, WMMA_N, WMMA_K, int8_t, wmma::row_major> a_frag;
    wmma::fragment<wmma::matrix_b, WMMA_M, WMMA_N, WMMA_K, int8_t, wmma::col_major> b_frag;
    wmma::fragment<wmma::accumulator, WMMA_M, WMMA_N, WMMA_K, int32_t> acc_frag;

    for (int64_t block_pos = blockIdx.x; ; block_pos += gridDim.x) {
        int64_t warp_pos = block_pos * blockDim.x/WARP_SIZE + warpId;
        int64_t WARP_M = warp_pos / (TILES_N * TILES_K);
        int64_t WARP_N = warp_pos % (TILES_N * TILES_K) / TILES_K;
        int64_t WARP_K = warp_pos % TILES_K;

        if (WARP_M >= TILES_M) {
            break;
        }

        int64_t aRow = WARP_M * WMMA_M;
        int64_t aCol = WARP_K * WMMA_K;
        wmma::load_matrix_sync(a_frag, x + aRow*GLOBAL_K + aCol, GLOBAL_K);

        int64_t bRow = WARP_N * WMMA_N;
        int64_t bCol = WARP_K * WMMA_K/8;
        for (int64_t i = laneId; i < WMMA_N * WMMA_K/8; i += WARP_SIZE) {
            int64_t THREAD_M = i / (WMMA_K/8);
            int64_t THREAD_K = i % (WMMA_K/8);

            int16_t weight_compressed = weights_compressed[(bRow+THREAD_M) * (GLOBAL_K/8) + (bCol+THREAD_K)];
            int16_t bits_abs = weight_compressed & ((1 << 8) - 1);
            int16_t bits_sign = (weight_compressed >> 8) & ((1 << 7) - 1);
            bool bit_shift = (weight_compressed >> 15) & ((1 << 1) - 1);

            int64_t packed = codebook_abs[bits_abs] ^ codebook_sign[bits_sign];
            packed -= bit_shift * 0x0202020202020202;
            packed |= 0x0101010101010101;

            decoded[i] = packed;   // little-endian
        }
        wmma::load_matrix_sync(b_frag, reinterpret_cast<int8_t *>(decoded), WMMA_K);

        wmma::fill_fragment(acc_frag, 0);

        wmma::mma_sync(acc_frag, a_frag, b_frag, acc_frag);

        wmma::store_matrix_sync(result, acc_frag, WMMA_N, wmma::mem_row_major);

        // __syncthreads();

        // TODO: aggregate result together, and call atomicAdd in the first warp

        int64_t cRow = WARP_M * WMMA_M;
        int64_t cCol = WARP_N * WMMA_N;
        for (int64_t i = laneId; i < WMMA_M * WMMA_N; i += WARP_SIZE) {
            int64_t cRowThread = cRow + i / WMMA_N;
            int64_t cColThread = cCol + i % WMMA_N;

            atomicAdd(output + cRowThread*GLOBAL_N + cColThread, result[i]);
        }
    }
}


__host__ torch::Tensor decode_matmul(
        torch::Tensor x,
        torch::Tensor weights_compressed,
        torch::Tensor codebook_abs,
        torch::Tensor codebook_sign) {
    CHECK_INPUT(x);
    CHECK_INPUT(weights_compressed);
    CHECK_INPUT(codebook_abs);
    CHECK_INPUT(codebook_sign);

    TORCH_CHECK(x.scalar_type() == torch::kInt8);
    TORCH_CHECK(weights_compressed.scalar_type() == torch::kInt16);
    TORCH_CHECK(codebook_abs.scalar_type() == torch::kInt64);
    TORCH_CHECK(codebook_sign.scalar_type() == torch::kInt64);
    TORCH_CHECK(x.size(-1) == weights_compressed.size(-1) << 3);

    int64_t GLOBAL_M = x.size(-2);
    int64_t GLOBAL_N = weights_compressed.size(-2);
    int64_t GLOBAL_K = x.size(-1);

    TORCH_CHECK(GLOBAL_M % WMMA_M == 0, "GLOBAL_M is not divisible by WMMA_M");
    TORCH_CHECK(GLOBAL_N % WMMA_N == 0, "GLOBAL_N is not divisible by WMMA_N");
    TORCH_CHECK(GLOBAL_K % WMMA_K == 0, "GLOBAL_K is not divisible by WMMA_K");

    at::DeviceGuard guard(x.device());
    torch::TensorOptions options = torch::TensorOptions()
        .dtype(torch::kInt32)
        .layout(torch::kStrided)
        .device(torch::kCUDA)
        .requires_grad(false);
    torch::Tensor output = torch::zeros(std::vector<int64_t>{GLOBAL_M, GLOBAL_N}, options);

    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, x.get_device());
    int64_t grid_size = 2 * static_cast<int64_t>(deviceProp.multiProcessorCount);
    int64_t block_size = 256;
    int64_t smem_size = (block_size/WARP_SIZE) * WMMA_N * (WMMA_K/8) * sizeof(int64_t);
    smem_size += (block_size/WARP_SIZE) * WMMA_M * WMMA_N * sizeof(int32_t);
    TORCH_CHECK_LE(smem_size, SMEM_SIZE_MAX);
    at::cuda::CUDAStream stream = at::cuda::getCurrentCUDAStream();

    TORCH_CHECK(GLOBAL_K % (WMMA_K * block_size/WARP_SIZE) == 0, "A block may cover multiple tiles in the result matrix");

    decode_matmul_kernel<<<grid_size, block_size, smem_size, stream>>>(
            output.data_ptr<int32_t>(),
            x.data_ptr<int8_t>(),
            weights_compressed.data_ptr<int16_t>(),
            codebook_abs.data_ptr<int64_t>(),
            codebook_sign.data_ptr<int64_t>(),
            GLOBAL_M,
            GLOBAL_N,
            GLOBAL_K);
    gpuErrchk(hipPeekAtLastError());
    gpuErrchk(hipDeviceSynchronize());

    return output;
}


PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("decode_matmul", &decode_matmul, "Fused Decode Matrix Multiplication");
}
