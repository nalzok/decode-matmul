#include "hip/hip_runtime.h"
#include <iostream>
#include <hip/hip_runtime.h>
#include <mma.h>

#include <ATen/ATen.h>
#include <ATen/Context.h>
#include <ATen/cuda/HIPContext.h>
#include <ATen/Dispatch.h>

#include <torch/types.h>
#include <torch/extension.h>

using namespace nvcuda;

#define CHECK_CUDA(x)           TORCH_CHECK(x.is_cuda(), #x " must be a CUDA tensor")
#define CHECK_CONTIGUOUS(x)     TORCH_CHECK(x.is_contiguous(), #x " must be contiguous")
#define CHECK_INPUT(x) 	        do { CHECK_CUDA(x); CHECK_CONTIGUOUS(x); } while(false)
#define gpuErrchk(ans)          do { gpuAssert((ans), __FILE__, __LINE__); } while (false)

#define WMMA_M                  8
#define WMMA_N                  32
#define WMMA_K                  16

#define WARP_SIZE               32

#define SMEM_SIZE_MAX           (48 * 1024)


__host__ static inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
    if (code != hipSuccess)
    {
        fprintf(stderr, "GPUassert[%s:%d]: %s\n", file, line, hipGetErrorString(code));
        if (abort) exit(code);
    }
}


__global__ static void decode_matmul_kernel(
        int32_t *__restrict__ output,
        const int8_t *__restrict__ x,
        const int16_t *__restrict__ weights_compressed,
        const int64_t *__restrict__ codebook_abs,
        const int64_t *__restrict__ codebook_sign,
        int64_t GLOBAL_M,
        int64_t GLOBAL_N,
        int64_t GLOBAL_K) {

    int64_t warpId = threadIdx.x / WARP_SIZE;
    int64_t laneId = threadIdx.x % WARP_SIZE;

    extern __shared__ __align__(sizeof(int64_t)) char decoded_block_raw[];
    int64_t *decoded_block = reinterpret_cast<int64_t *>(decoded_block_raw);
    int64_t *decoded = decoded_block + warpId * WMMA_N * WMMA_K / 8;

    int64_t TILES_M = GLOBAL_M / WMMA_M;
    int64_t TILES_N = GLOBAL_N / WMMA_N;

    wmma::fragment<wmma::matrix_a, WMMA_M, WMMA_N, WMMA_K, int8_t, wmma::row_major> a_frag;
    wmma::fragment<wmma::matrix_b, WMMA_M, WMMA_N, WMMA_K, int8_t, wmma::col_major> b_frag;
    wmma::fragment<wmma::accumulator, WMMA_M, WMMA_N, WMMA_K, int32_t> acc_frag;

    for (int64_t block_pos = blockIdx.x; ; block_pos += gridDim.x) {
        int64_t warp_pos = block_pos / WARP_SIZE + warpId;
        int64_t WARP_M = warp_pos / TILES_N;
        int64_t WARP_N = warp_pos % TILES_N;

        if (WARP_M >= TILES_M) {
            break;
        }

        wmma::fill_fragment(acc_frag, 0);

        for (int64_t k = 0; k < GLOBAL_K; k += WMMA_K) {
            int64_t aRow = WARP_M * WMMA_M;
            int64_t aCol = k;

            wmma::load_matrix_sync(a_frag, x + aRow*GLOBAL_K + aCol, GLOBAL_K);

            int64_t bRow = WARP_N * WMMA_N;
            int64_t bCol = k / 8;

            for (int64_t i = laneId; i < WMMA_N * WMMA_K/8; i += WARP_SIZE) {
                int64_t THREAD_M = i / (WMMA_K/8);
                int64_t THREAD_K = i % (WMMA_K/8);

                int16_t weight_compressed = weights_compressed[(bRow + THREAD_M) * (GLOBAL_K/8) + bCol + THREAD_K];
                int16_t bits_abs = weight_compressed & ((1 << 8) - 1);
                int16_t bits_sign = (weight_compressed >> 8) & ((1 << 7) - 1);
                bool bit_shift = (weight_compressed >> 15) & ((1 << 1) - 1);

                int64_t packed = codebook_abs[bits_abs] ^ codebook_sign[bits_sign];
                packed -= bit_shift * 0x0202020202020202;
                packed |= 0x0101010101010101;

                decoded[THREAD_M*(WMMA_K/8) + THREAD_K] = packed;   // little-endian
            }

            wmma::load_matrix_sync(b_frag, reinterpret_cast<int8_t *>(decoded), WMMA_K);

            wmma::mma_sync(acc_frag, a_frag, b_frag, acc_frag);
        }

        int64_t cRow = WARP_M * WMMA_M;
        int64_t cCol = WARP_N * WMMA_N;

        wmma::store_matrix_sync(output + cRow*GLOBAL_N + cCol, acc_frag, GLOBAL_N, wmma::mem_row_major);
    }
}


__host__ torch::Tensor decode_matmul(
        torch::Tensor x,
        torch::Tensor weights_compressed,
        torch::Tensor codebook_abs,
        torch::Tensor codebook_sign) {
    CHECK_INPUT(x);
    CHECK_INPUT(weights_compressed);
    CHECK_INPUT(codebook_abs);
    CHECK_INPUT(codebook_sign);

    TORCH_CHECK(x.scalar_type() == torch::kInt8);
    TORCH_CHECK(weights_compressed.scalar_type() == torch::kInt16);
    TORCH_CHECK(codebook_abs.scalar_type() == torch::kInt64);
    TORCH_CHECK(codebook_sign.scalar_type() == torch::kInt64);
    TORCH_CHECK(x.size(-1) == weights_compressed.size(-1) << 3);

    int64_t GLOBAL_M = x.size(-2);
    int64_t GLOBAL_N = weights_compressed.size(-2);
    int64_t GLOBAL_K = x.size(-1);

    TORCH_CHECK(GLOBAL_M % WMMA_M == 0, "GLOBAL_M is not divisible by WMMA_M");
    TORCH_CHECK(GLOBAL_N % WMMA_N == 0, "GLOBAL_N is not divisible by WMMA_N");
    TORCH_CHECK(GLOBAL_K % WMMA_K == 0, "GLOBAL_K is not divisible by WMMA_K");

    at::DeviceGuard guard(x.device());
    torch::TensorOptions options = torch::TensorOptions()
        .dtype(torch::kInt32)
        .layout(torch::kStrided)
        .device(torch::kCUDA)
        .requires_grad(false);
    torch::Tensor output = torch::zeros(std::vector<int64_t>{GLOBAL_M, GLOBAL_N}, options);

    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, x.get_device());
    int64_t block_size = 256;
    int64_t smem_size = (block_size / WARP_SIZE) * WMMA_N * (WMMA_K / 8) * sizeof(int64_t);
    TORCH_CHECK_LE(smem_size, SMEM_SIZE_MAX);
    at::cuda::CUDAStream stream = at::cuda::getCurrentCUDAStream();

    decode_matmul_kernel<<<deviceProp.multiProcessorCount, block_size, smem_size, stream>>>(
            output.data_ptr<int32_t>(),
            x.data_ptr<int8_t>(),
            weights_compressed.data_ptr<int16_t>(),
            codebook_abs.data_ptr<int64_t>(),
            codebook_sign.data_ptr<int64_t>(),
            GLOBAL_M,
            GLOBAL_N,
            GLOBAL_K);
    gpuErrchk(hipPeekAtLastError());
    gpuErrchk(hipDeviceSynchronize());

    return output;
}


PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("decode_matmul", &decode_matmul, "Fused Decode Matrix Multiplication");
}
